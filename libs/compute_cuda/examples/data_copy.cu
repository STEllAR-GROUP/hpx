//  Copyright (c) 2016 Hartmut Kaiser
//
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <hpx/include/compute.hpp>
#include <hpx/include/parallel_copy.hpp>

#include <hpx/hpx_init.hpp>

#include <iostream>
#include <numeric>
#include <random>
#include <string>
#include <vector>

int hpx_main(hpx::program_options::variables_map& vm)
{
    auto seed = std::random_device{}();
    if (vm.count("seed"))
        seed = vm["seed"].as<unsigned int>();

    std::cout << "using seed: " << seed << std::endl;
    std::mt19937 gen(seed);
    std::uniform_int_distribution<> dis(2, 101);

    // create data vector on host
    int const N = 100;
    std::vector<int> h_A(N);
    std::vector<int> h_B(N);
    std::iota(h_A.begin(), h_A.end(), dis(gen));

    hpx::compute::cuda::target target;

    // create data vector on device
    typedef hpx::compute::cuda::allocator<int> allocator_type;
    allocator_type alloc(target);

    hpx::compute::vector<int, allocator_type> d_A(N, alloc);

    hpx::future<void> f = target.get_future();

    f.get();

    // copy data from host to device
    hpx::parallel::copy(
        hpx::parallel::execution::par,
        h_A.begin(), h_A.end(), d_A.begin());

    // copy data from device to host
    hpx::parallel::copy(
        hpx::parallel::execution::par,
        d_A.begin(), d_A.end(), h_B.begin());

    if(std::equal(h_A.begin(), h_A.end(), h_B.begin()))
        std::cout << "Copy succeeded!\n";
    else
        std::cout << "Copy not successful :(\n";

    return hpx::finalize();
}

int main(int argc, char* argv[])
{
    // add command line option which controls the random number generator seed
    using namespace hpx::program_options;
    options_description desc_commandline(
        "Usage: " HPX_APPLICATION_STRING " [options]");

    desc_commandline.add_options()
        ("seed,s", value<unsigned int>(),
        "the random number generator seed to use for this run")
        ;

    // Initialize and run HPX
    return hpx::init(desc_commandline, argc, argv);
}
