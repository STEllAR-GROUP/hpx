//  Copyright (c) 2018 John Biddiscombe
//
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include "hip/hip_runtime.h"

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

